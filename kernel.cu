#include "hip/hip_runtime.h"
/*
    MIT License
    Author: Fred Kyung-jin Rezeau <fred@litemint.com>, 2024
    Permission is granted to use, copy, modify, and distribute this software for any purpose
    with or without fee.
    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <cstring>
#include <cstddef>

#include "utils/keccak.cuh"

#define CUDA_CALL(call)                                                \
    do {                                                               \
        hipError_t err = call;                                        \
        if (err != hipSuccess) {                                      \
            fprintf(stderr, "CUDA Error in %s, line %d: %s\n",         \
                    __FILE__, __LINE__, hipGetErrorString(err));      \
            exit(EXIT_FAILURE);                                        \
        }                                                              \
    } while (0)

__device__ void updateNonce(std::uint64_t val, std::uint8_t* buffer) {
    // Xdr bytes first.
    buffer[0] = 0;
    buffer[1] = 0;
    buffer[2] = 0;
    buffer[3] = 5;
    for (int i = 4; i < 12; i++) {
        buffer[11 - (i - 4)] = static_cast<std::uint8_t>(val & 0xFF);
        val >>= 8;
    }
}

__device__ bool check(const std::uint8_t* hash, int difficulty) {
    int zeros = 0;
    for (int i = 0; i < 32; ++i) {
        zeros += (hash[i] == 0) ? 2 : ((hash[i] >> 4) == 0 ? 1 : 0);
        if (hash[i] != 0 || zeros >= difficulty)
            break;
    }
    return zeros >= difficulty;
}

__global__ void run(std::uint8_t* data, int dataSize, std::uint64_t startNonce,
                                 int nonceOffset, std::uint64_t batchSize, int difficulty,
                                 int* found, std::uint8_t* output, std::uint64_t* validNonce) {
    std::uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    std::uint64_t stride = gridDim.x * blockDim.x;
    if (idx >= batchSize || atomicAdd(found, 0) == 1)
        return;
    std::uint64_t nonceEnd = startNonce + batchSize;
    const int maxSize = 256;

    // Nonce distribution is based on thread id - spaced by stride.
    for (std::uint64_t nonce = startNonce + idx; nonce < nonceEnd; nonce += stride) {
        std::uint8_t threadData[maxSize];
        if (dataSize > maxSize)
            return;
        for (int i = 0; i < dataSize; i++) {
            threadData[i] = data[i];
        }
        updateNonce(nonce, &threadData[nonceOffset]);
        std::uint8_t hash[32];
        keccak256(threadData, dataSize, hash);
        if (check(hash, difficulty)) {
            if (atomicCAS(found, 0, 1) == 0) {
                memcpy(output, hash, 32);
                atomicExch(reinterpret_cast<unsigned long long int*>(validNonce), nonce);
            }
            return;
        }
        if (atomicAdd(found, 0) == 1)
            return;
    }
}

extern "C" int executeKernel(std::uint8_t* data, int dataSize, std::uint64_t startNonce, int nonceOffset, std::uint64_t batchSize,
    int difficulty, int threadsPerBlock, std::uint8_t* output, std::uint64_t* validNonce) {
    std::uint8_t* deviceData;
    std::uint8_t* deviceOutput;
    std::size_t outputSize = 32 * sizeof(std::uint8_t);
    int found = 0;
    int* deviceFound;
    std::uint64_t* deviceNonce;
    hipDeviceProp_t deviceProp;
    CUDA_CALL(hipGetDeviceProperties(&deviceProp, 0));
    CUDA_CALL(hipMalloc((void**)&deviceFound, sizeof(int)));
    CUDA_CALL(hipMemcpy(deviceFound, &found, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void**)&deviceData, dataSize));
    CUDA_CALL(hipMalloc((void**)&deviceOutput, outputSize));
    CUDA_CALL(hipMalloc((void**)&deviceNonce, sizeof(std::uint64_t)));
    CUDA_CALL(hipMemset(deviceNonce, 0, sizeof(std::uint64_t)));
    CUDA_CALL(hipMemcpy(deviceData, data, dataSize, hipMemcpyHostToDevice));

    int threads = threadsPerBlock;
    std::uint64_t blocks = (batchSize + threads - 1) / threads;
    if (blocks > deviceProp.maxGridSize[0]) {
        blocks = deviceProp.maxGridSize[0];
    }
    std::uint64_t adjustedBatchSize = blocks * threads;
    run<<<(unsigned int)blocks, threads>>>(deviceData, dataSize, startNonce,
        nonceOffset, adjustedBatchSize, difficulty, deviceFound, deviceOutput, deviceNonce);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(output, deviceOutput, outputSize, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(&found, deviceFound, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(validNonce, deviceNonce, sizeof(std::uint64_t), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(deviceData));
    CUDA_CALL(hipFree(deviceOutput));
    CUDA_CALL(hipFree(deviceFound));
    CUDA_CALL(hipFree(deviceNonce));
    return found;
}
